#include "CUDA_Array.cuh"

// Constructors:

// Empty constructor
template <typename ITYPE, typename RTYPE>
CUDA_Array<ITYPE, RTYPE>::CUDA_Array()
{
    Array<ITYPE, RTYPE>::arrSize = 0;
    d_data = nullptr;
}

// Parametrized constructor, extends the base class constructor
template <typename ITYPE, typename RTYPE>
CUDA_Array<ITYPE, RTYPE>::CUDA_Array(ITYPE& n)
{
    Array<ITYPE, RTYPE>::arrSize = n;
    // Allocate device memory for the data
    CUDA_CHECK(hipMalloc((void**)&d_data, n * sizeof(RTYPE)));
}

// Destructor
template <typename ITYPE, typename RTYPE>
CUDA_Array<ITYPE, RTYPE>::~CUDA_Array()
{
    // Free device memory
    if (d_data != nullptr) {
        CUDA_CHECK(hipFree(d_data));
        d_data = nullptr;
    }
}

// Getters:

// New implementation of data getter to account for device memory
template <typename ITYPE, typename RTYPE>
RTYPE* CUDA_Array<ITYPE, RTYPE>::data()
{
    // Since is called from Host code, need to copy data from device to host
    RTYPE* h_data = (RTYPE*)calloc(Array<ITYPE, RTYPE>::arrSize, sizeof(RTYPE));
    CUDA_CHECK(hipMemcpy(h_data, d_data, Array<ITYPE, RTYPE>::arrSize * sizeof(RTYPE), hipMemcpyDeviceToHost));
    return h_data; // Return host pointer
}

// Setters:

// New setSize method to allocate device memory
template <typename ITYPE, typename RTYPE>
void CUDA_Array<ITYPE, RTYPE>::setSize(ITYPE& n)
{
    // Check that the array is empty before allocating new memory
    if (this->arrSize != 0) {
        this->errorCode = 5;
        this->handleError(this->errorCode);
    }

    // Sert the new size
    if (n <= 0)
    {
        this->errorCode = 1;
        this->handleError(this->errorCode);
    }
    this->arrSize = n;

    // Allocate device memory for the data
    CUDA_CHECK(hipMalloc((void**)&d_data, n * sizeof(RTYPE)));
}

// Set data now copies from host to device
template <typename ITYPE, typename RTYPE>
void CUDA_Array<ITYPE, RTYPE>::setData(ITYPE& n, RTYPE* data)
{
    // Check that the array is not empty
    if (this->arrSize == 0) {
        this->errorCode = 6;
        this->handleError(this->errorCode);
    }
    if (n != this->arrSize) {
        this->errorCode = 2;
        this->handleError(this->errorCode);
    }

    // Set the data with host2device copy
    CUDA_CHECK(hipMemcpy(d_data, data, n * sizeof(RTYPE), hipMemcpyHostToDevice));
}

// Define specific implementations
template class CUDA_Array<uint32_t, float>;
template class CUDA_Array<uint64_t, float>;
template class CUDA_Array<uint32_t, double>;
template class CUDA_Array<uint64_t, double>;